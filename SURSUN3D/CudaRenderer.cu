#include "hip/hip_runtime.h"

#include "CudaTools.h"
#include "CudaRenderer.h"
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>
#include <cutil_math.h>
//texture<float4, hipTextureType2D, hipReadModeElementType> texMapVertices; 
//texture<float4, hipTextureType2D, hipReadModeElementType> texMapNormals;  
//texture<float4, hipTextureType1D, hipReadModeElementType> texGeometryVertices; 
TriangleMesh* g_pGeometryNodes = NULL;
unsigned int* g_pValidNodes = NULL;
unsigned int* g_pValidNodes_Host = NULL;

bool* g_pValidNodeMasks = NULL;
bool* g_pValidNodeMasks_Host = NULL;
unsigned int m_numofnodes;
unsigned int m_numofvalidnodes;
//hipArray* g_pGeometryVertices;
float3* g_pGeometryVertices = NULL;
float3* g_pLights = NULL;
float3* g_pLights_host = NULL;
float* g_pViewMatrices = NULL;
bool* g_pNodeCullInfo = NULL;
unsigned int m_numoflights;
Box* g_pFrustrums = NULL;
TriangleMesh* g_pTargetGeometry = NULL;
TriangleMesh g_pTargetGeometry_Host;
bool* g_pExternalValidNodeMasks = NULL;
void setExternalValidNodeMasks(bool*& masks)
{
	g_pExternalValidNodeMasks = masks;
}

void executeKernelFrustrumCullingCPU(int width, int height,float3 light,Box frustrum,float* matView,TriangleMesh* geometryNodes,unsigned int numofnodes,TriangleMesh* targetGeometry)
{
 


	 unsigned int curoffset = 0;
	 unsigned int* pValidNodes = g_pValidNodes_Host;
	 m_numofvalidnodes = 0;
	 bool* pExternalMask = g_pExternalValidNodeMasks;

	 //for(bool* pMask=g_pValidNodeMasks_Host;pMask<g_pValidNodeMasks_Host+m_numofnodes;pMask++)
	 //{

		// if(*pMask == false && (pExternalMask && *pExternalMask != false))
		// {
		//	 m_numofvalidnodes++;
		//	 *pValidNodes = curoffset;pValidNodes++;
		// }	
		// if(pExternalMask)
		//    pExternalMask++;
		// curoffset++;
	 //}



	unsigned int cullcount = 0;
   for(int i=0;i<numofnodes;i++)
   {
	   TriangleMesh node = geometryNodes[i];
	   Sphere sp;
	   sp.Radius = node.BBRadius;
	   sp.Center = vectorMulMat(node.BBCenter,matView);
	    bool isCulled  = 0;
	      float angle = asinf(light.z);
		//�������Ƕȱ�����Ӱ���߳�,70.0=1.22173,80=1.4835298;89=1.553341722222222
	   if(angle > 1.4835298)
		   angle = 1.4835298;

	   float shadowD = targetGeometry->ZMax / tan(angle); 
		//��ӰͶ�����С��������֮��ľ�����ϰ�Χ�а뾶������������ӰͶ�����
	   float dist = length(targetGeometry->BBCenter - node.BBCenter);
/*	   if( dist - targetGeometry->BBRadius - node.BBRadius > shadowD)
	   {
		      isCulled = 1;
			  cullcount++;
	   }
	   else*/ 
	   if(!intersects(frustrum,sp))
	   {
			   isCulled = 1;
			   cullcount++;
	   }
	   else
	   {
			 m_numofvalidnodes++;
			 *pValidNodes = curoffset;pValidNodes++;
	   }
	   if(pExternalMask)
		   pExternalMask++;
	   curoffset++;
	   geometryNodes[i] = node;

   }
   hipMemcpy( g_pValidNodes,g_pValidNodes_Host,sizeof(unsigned int) * m_numofvalidnodes ,hipMemcpyHostToDevice);
   printf("%d/%d\n",cullcount,numofnodes);

} 



__global__ void kernelFrustrumCulling(int width, int height,float3 light,unsigned int lightindex,Box* frustrum,float* matView,TriangleMesh* geometryNodes,bool* masks,unsigned int numofnodes,TriangleMesh* targetGeometry)
{
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; // row number 
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; // col number

   unsigned int index = i*width+j;
   if(index >= numofnodes)
	   return;	
   TriangleMesh node = geometryNodes[index];
   Sphere sp;
   sp.Radius = node.BBRadius;
   bool isCulled = 0;
   //float3 light = lights[ilight];
   //����ӰͶ�䳤��
   float angle = asinf(light.z);
	//�������Ƕȱ�����Ӱ���߳�,70.0=1.22173,80=1.4835298;89=1.553341722222222
   if(angle > 1.4835298)
	   angle = 1.4835298;

   float shadowD = targetGeometry->ZMax / tan(angle); 
	//��ӰͶ�����С��������֮��ľ�����ϰ�Χ�а뾶������������ӰͶ�����
   float dist = length(targetGeometry->BBCenter - node.BBCenter);
   //if( dist - targetGeometry->BBRadius - node.BBRadius > shadowD)
   //{
   //     isCulled = 1;
   //}
   //else
   //{
	   sp.Center = vectorMulMat(node.BBCenter,matView + lightindex*16);
	   if(!intersects(frustrum[lightindex],sp))
		   isCulled = 1;
   //}

  //if( node.ID == targetGeometry->ID)
	 // isCulled = 0;
   
   masks[index]  = isCulled;
} 

bool* getNodeMasks()
{
   return g_pValidNodeMasks_Host;
}

void executeKernelFrustrumCulling(unsigned int lightnum)
{
	 hipError_t error;
	 dim3 gridDim3,blockDim3;
	 unsigned int height,width;
	 height = 16;
	 width  = m_numofnodes / height + 1;
     computeKernelDims(height,width,gridDim3,blockDim3);
 	 kernelFrustrumCulling<<< gridDim3, blockDim3 >>>(   
width, 
height,
g_pLights_host[lightnum],
lightnum,
g_pFrustrums, 
g_pViewMatrices,
g_pGeometryNodes,
g_pValidNodeMasks,
m_numofnodes,
g_pTargetGeometry);
 	 hipDeviceSynchronize();

	 m_numofvalidnodes = 0;
     hipMemcpy(g_pValidNodeMasks_Host,g_pValidNodeMasks,sizeof(bool) * m_numofnodes,hipMemcpyDeviceToHost);
	 unsigned int curoffset = 0;
	 unsigned int* pValidNodes = g_pValidNodes_Host;
	 m_numofvalidnodes = 0;
	 bool* pExternalMask = g_pExternalValidNodeMasks;

	 for(bool* pMask=g_pValidNodeMasks_Host;pMask<g_pValidNodeMasks_Host+m_numofnodes;pMask++)
	 {

		 if(*pMask == false)
		 {
			 if(!pExternalMask || *pExternalMask != false)
			 {
			     m_numofvalidnodes++;
			    *pValidNodes = curoffset;pValidNodes++;
			 }
		 }	
		 if(pExternalMask)
		    pExternalMask++;
		 curoffset++;
	 }
    hipMemcpy( g_pValidNodes,g_pValidNodes_Host,sizeof(unsigned int) * m_numofvalidnodes ,hipMemcpyHostToDevice);

	//printf("remaining: %d/%d\n",m_numofvalidnodes,m_numofnodes);

}
void setTargetGeometry(TriangleMesh* targetGeometry)
{
	 g_pTargetGeometry_Host = *targetGeometry;
	 size_t sizeInBytes = sizeof(TriangleMesh);
     hipMemcpy(g_pTargetGeometry,targetGeometry,sizeInBytes,hipMemcpyHostToDevice);
}
void setGeometry(float3* vertices,unsigned int numofvertices,TriangleMesh* geometryNodes,unsigned int numofnodes)
{
	hipError_t error;
	size_t sizeInBytes ;
    sizeInBytes = numofvertices * sizeof(float3);
	//hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32,hipChannelFormatKindFloat);
 //   hipMallocArray( &g_pGeometryVertices, &channelDesc, numofvertices, 1 ); 
 //   hipMemcpyToArray( g_pGeometryVertices, 0, 0, vertices, numofvertices, hipMemcpyHostToDevice);


    error  = hipMalloc((void**)&g_pGeometryVertices, sizeInBytes);
	hipMemcpy(g_pGeometryVertices,vertices,sizeInBytes,hipMemcpyHostToDevice);
    //// set texture parameters
    //texGeometryVertices.addressMode[0] = hipAddressModeWrap;
    //texGeometryVertices.addressMode[1] = hipAddressModeWrap;
    //texGeometryVertices.filterMode = hipFilterModePoint;
    //texGeometryVertices.normalized = false;    
    //cutilSafeCall(hipBindTextureToArray(texGeometryVertices, g_pGeometryVertices, channelDesc));

	 m_numofnodes = numofnodes;
     sizeInBytes = m_numofnodes * sizeof(TriangleMesh);
     error  = hipMalloc((void**)&g_pGeometryNodes, sizeInBytes);
     hipMemcpy(g_pGeometryNodes,geometryNodes,sizeInBytes,hipMemcpyHostToDevice);

     error  = hipMalloc((void**)&g_pValidNodes, sizeof(unsigned int) * m_numofnodes);
     g_pValidNodes_Host  = (unsigned int*)malloc(sizeof(unsigned int) * m_numofnodes);
	 
     error  = hipMalloc((void**)&g_pValidNodeMasks, sizeof(bool) * m_numofnodes);
     g_pValidNodeMasks_Host  = (bool*)malloc(sizeof(bool) * m_numofnodes);



     sizeInBytes = sizeof(TriangleMesh);
     error  = hipMalloc((void**)&g_pTargetGeometry, sizeInBytes);

}

void updateGeometry(float3* vertices,unsigned int numofvertices,TriangleMesh* geometryNodes,unsigned int numofnodes)
{
	hipError_t error;
	size_t sizeInBytes;
    sizeInBytes = numofvertices * sizeof(float3);
	hipMemcpy(g_pGeometryVertices,vertices,sizeInBytes,hipMemcpyHostToDevice);

	 m_numofnodes = numofnodes;
     sizeInBytes = m_numofnodes * sizeof(TriangleMesh);
     hipMemcpy(g_pGeometryNodes,geometryNodes,sizeInBytes,hipMemcpyHostToDevice);
}
//void setGeometryNode(TriangleMesh* node,unsigned int index)
//{
//     hipMemcpy(g_pGeometryNodes+index,node,sizeof(TriangleMesh),hipMemcpyHostToDevice);
//}





////float3* g_pLights;
////float* g_pViewMatrices;
////int* g_pNodeCullInfo;

void setLights(float3* lights,unsigned int numoflights)
{
     
	if(g_pViewMatrices)
       hipFree(g_pViewMatrices);
	g_pViewMatrices = NULL;

	if(g_pLights)
       hipFree(g_pLights);
	g_pLights = NULL;

	if(g_pFrustrums)
       hipFree(g_pFrustrums);
	g_pFrustrums = NULL;


	 size_t sizeInBytes;
	 hipError_t error;
	 m_numoflights = numoflights;
     sizeInBytes = 16 * sizeof(float) * numoflights;
     error  = hipMalloc((void**)&g_pViewMatrices, sizeInBytes);
     //hipMemcpy(g_pViewMatrices,viewMatrices,sizeInBytes,hipMemcpyHostToDevice);

     sizeInBytes = numoflights * sizeof(float3);
     error  = hipMalloc((void**)&g_pLights, sizeInBytes);
     hipMemcpy(g_pLights,lights,sizeInBytes,hipMemcpyHostToDevice);
	 //g_pLights = lights;//(float3*)malloc(numoflights * sizeof(float3));
	 //memcpy(g_pLights,lights,numoflights * sizeof(float3));
     g_pLights_host = lights;


	 sizeInBytes = sizeof(Box) * m_numoflights;
     error  = hipMalloc((void**)&g_pFrustrums, sizeInBytes);

}


__device__ unsigned int cast_Ray(unsigned int index,float3 lightDir,
									 float4 pos4,float4 normal4,
									 TriangleMesh* geometryNodes,unsigned int numofnodes,
									 unsigned int* validNodes,unsigned int numofvalidnodes,
									 float3* geometryVertices)
{

   float3 pos = make_float3(pos4);
   if(normal4.w < -1)
   {
	   return 0;
   }

   float3 normal = make_float3(normal4);
   //unsigned int faceindex = normal4.w;
   //float3 f0 = geometryVertices[faceindex*3];
   //float3 f1 = geometryVertices[faceindex*3 + 1];
   //float3 f2 = geometryVertices[faceindex*3 + 2];

   if(isParallel(lightDir,normal)) 
   {  
	   return 1;
   }

   float b = dot(normal, lightDir);  
   if (b <= 0)
   {
        return 1;
   }

   int isShadowed = 0;    
   for(int k=0;k<numofvalidnodes;k++)
   {	
	   TriangleMesh node = geometryNodes[validNodes[k]]; 
	   for(unsigned int l=0;l<node.FaceCount;l++)
	   {

	     //  if((unsigned int)normal4.w ==  node.FaceStart + l)
			   //continue;
           float3 v0,v1,v2;

		   v0 = geometryVertices[(node.FaceStart + l) * 3 + 0];
		   v1 = geometryVertices[(node.FaceStart + l) * 3 + 1];
		   v2 = geometryVertices[(node.FaceStart + l) * 3 + 2];
		   float3 n = cross(v1 - v0, v2 - v0);
		   b = dot(n, lightDir);  

		   if (b <= 0)
		   {
				continue;
		   }
	
		   float dist = intersectTriangle2(pos, lightDir, v0,v1,v2);
		   if(dist > 0.01 && dist < 1000000)
		   {
               isShadowed = 1;//node.ID + 2;
			   break;
		   }


	   }
	   if(isShadowed > 0)
	   {
		   break;
	   }	 		
   }

   return isShadowed;
    
} 


__global__ void kernelShadowCasting(bool* output, int width, int height, unsigned int startIndex,unsigned int offset,float3* lightDir,unsigned int lightindex,
									 float4* mapVertices,float4* mapNormals,
									 TriangleMesh* geometryNodes,unsigned int numofnodes,
									 unsigned int* validNodes,unsigned int numofvalidnodes,
									 float3* geometryVertices)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // row number 
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col number

    if(i >= height || j >= width)
		return;

   unsigned int index = i*width+j + startIndex;
   float4 pos4 = mapVertices[index];
   float4 normal4 = mapNormals[index];
   int shadow = cast_Ray(index,lightDir[lightindex],pos4,normal4,geometryNodes,numofnodes,validNodes,numofvalidnodes,geometryVertices);
   if(shadow > 0)
   {
       output[index + offset] = true;
   }
   else
   {
	   output[index + offset] = false;
   }

} 

void printfFloat3(float3 v)
{
	printf("(%f,%f,%f)\n",v.x,v.y,v.z);
}

void executeKernelShadowCasting(bool* output,float4* map_vertices,float4* map_normals,
				   int width, int height,float* matViews,Box* frustrums)
{   
	 size_t sizeInBytes;
     size_t grid_pitch;
	 size_t count = width * height;
     hipError_t error;
     //hipChannelFormatDesc channeldesc = hipCreateChannelDesc<float4>();
     float4* mapVertices_device;
     float4* mapNormals_device;
     sizeInBytes = sizeof(float4)*width*height;
     error = hipMalloc((void **)&mapVertices_device, sizeInBytes);
     hipMemcpy(mapVertices_device,map_vertices,sizeInBytes,hipMemcpyHostToDevice);

     error = hipMalloc((void **)&mapNormals_device, sizeInBytes);
     hipMemcpy(mapNormals_device,map_normals,sizeInBytes,hipMemcpyHostToDevice);


     sizeInBytes = 16 * sizeof(float) * m_numoflights;
     hipMemcpy(g_pViewMatrices,matViews,sizeInBytes,hipMemcpyHostToDevice);


     sizeInBytes = sizeof(Box) * m_numoflights;
     hipMemcpy(g_pFrustrums,frustrums,sizeInBytes,hipMemcpyHostToDevice);

	 //executeKernelFrustrumCulling();
	 bool* output_device;
     sizeInBytes = sizeof(bool)*width*height*m_numoflights;
     error = hipMalloc(&output_device, sizeInBytes);
	 dim3 gridDim3,blockDim3;



	 for(int i=0;i<m_numoflights;i++)
	 {
		 unsigned int startIndex = 0;
		 int rowstep = 100;
		 int currows = 0;
		 int rows;
	     unsigned int offset = i * height * width;
		 executeKernelFrustrumCulling(i);
		 while(startIndex < height * width)
		 {
			 rows = rowstep;
			 rows = min(rows,height-currows);

			 computeKernelDims(rows,width,gridDim3,blockDim3);
			 kernelShadowCasting<<< gridDim3, blockDim3 >>>
				(   
		output_device,  
		width, 
		rows,
		startIndex,
		offset,
		g_pLights,
		i,
		mapVertices_device,
		mapNormals_device,
		g_pGeometryNodes,
		m_numofnodes,
		g_pValidNodes,
		m_numofvalidnodes,
		g_pGeometryVertices
				);
			 startIndex+=rows*width;
			 currows+=rows;
			 error = hipDeviceSynchronize();	
			 //printf ("Cuda error -- %s\n", hipGetErrorString(hipGetLastError()));
	 }
	 }
 	


     hipMemcpy(output,output_device,sizeInBytes,hipMemcpyDeviceToHost);
	 hipFree(mapVertices_device);
	 hipFree(mapNormals_device);
	 hipFree(output_device);

}


void destroyGeometry()
{
    //hipUnbindTexture(texGeometryVertices);
	if(g_pGeometryVertices)
       hipFree(g_pGeometryVertices);
	g_pGeometryVertices = NULL;
	if(g_pViewMatrices)
       hipFree(g_pViewMatrices);
	g_pViewMatrices = NULL;
	if(g_pFrustrums)
       hipFree(g_pFrustrums);
	g_pFrustrums = NULL;
	if(g_pTargetGeometry)
       hipFree(g_pTargetGeometry);
	g_pTargetGeometry = NULL;
	if(g_pGeometryNodes)
       hipFree(g_pGeometryNodes);
	g_pGeometryNodes = NULL;
	if(g_pValidNodes)
        hipFree(g_pValidNodes);
	g_pValidNodes = NULL;
	if(g_pValidNodeMasks)
       hipFree(g_pValidNodeMasks);
	g_pValidNodeMasks = NULL;
	if(g_pValidNodes_Host)
       free(g_pValidNodes_Host);
	g_pValidNodes_Host = NULL;
	if(g_pValidNodeMasks_Host)
       free(g_pValidNodeMasks_Host); 
	g_pValidNodeMasks_Host = NULL;

	if(g_pLights)
       hipFree(g_pLights);
	g_pLights = NULL;

}
